#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>

#include <hip/hip_runtime.h>

#include <hipblas.h>
#include <assert.h>

// #include "dbg.h"

const int M = 1024;
const int K = 2048;
const int N = 1024;

const int BLOCK_SIZE = 16;

void initial(float* array, int size) {
    for (int i = 0; i < size; i++) {
        array[i] = (float)(rand() % 10 + 1);
    }
}

void printMatrix(float* array, int row, int col) {
    float* p = array;
    for (int i = 0; i < row; i++) {
        for (int j = 0; j < col; j++) {
            printf("%10lf", p[j]);
        }
        p = p + col;
        printf("\n");
    }
}

// array_A [M_p, K_p]   array_B [K_p, N_p]
void multiMatrixOnHost(float* array_A, float* array_B, float* array_C, int M_p, int K_p, int N_p) {
    for (int i = 0; i < M_p; i++) {
        for (int j = 0; j < N_p; j ++) {  // 计算array_C 的第[i, j]元素
            float sum_tmp = 0;
            for (int k = 0; k < K_p; k++) {
                sum_tmp += array_A[i * K_p + k] * array_B[k * N_p + j];
            }
            array_C[i * N_p + j] = sum_tmp;
        }
    }
}

// 不使用共享内存的版本
__global__ void multiMatrixOnDevice(float* array_A, float* array_B, float* array_C, int M_p, int K_p, int N_p) {
    // 每个线程计算矩阵C中的一个元素
    int row = threadIdx.x + blockDim.x * blockIdx.x;  // row 行数
    int col = threadIdx.y + blockDim.y * blockIdx.y;  // col 列数
    // printf("col:%d, row:%d\n", col, row);
    // printf("N_p:%d, M_p:%d\n", N_p, M_p);
    if (col < N_p && row < M_p) {
        float tmp = 0;
        for (int k = 0; k < K_p; k++) {
            tmp += array_A[row * K_p + k] * array_B[k * N_p + col];
            // 计算一次乘加需要读array_A/B各一次，读取global memory需要几百个cycle，而计算只需几个cycle
        }
        array_C[row * N_p + col] = tmp;
        // if (col == 2)
        //     printf("col:%d, row:%d, val:%f\n", col, row, tmp);
    }
}

__global__ void matrixMultiShared1(float* A, float* B, float* C, int M, int N, int K) {
    __shared__ float subA[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float subB[BLOCK_SIZE][BLOCK_SIZE];
    assert(blockDim.x == BLOCK_SIZE);

    int col = blockIdx.y * blockDim.y + threadIdx.y;  // 列号
    int row = blockIdx.x * blockDim.x + threadIdx.x;  // 行号

    float Csub = 0.0;
    // 通过for循环依次把numAcols/BLOCK_SIZE个子矩阵放入共享内存的subA, subB
    // 共享内存on-chip中的读约几十个cycle
    // 降低计算访问内存比
    // Loop over all the sub-matrices of A and B that are required to compute Csub
    // Multiply each pair of sub-matrices together and accumulate the results
    for (int i = 0; i < (K / BLOCK_SIZE); i++) {
        subA[threadIdx.x][threadIdx.y] = A[row * K + i * BLOCK_SIZE + threadIdx.y];
        subB[threadIdx.x][threadIdx.y] = B[(i * BLOCK_SIZE + threadIdx.x) * N + col];

        // synchronize to make sure the sub-matrieces are loaded before starting the computation
        __syncthreads();

        //计算每个元素
        for (int j = 0; j < BLOCK_SIZE; j++) {
            Csub = Csub + subA[threadIdx.x][j] * subB[j][threadIdx.y];
        }
        // synchronize to make sure that the preceding computation is done before starting the computation
        __syncthreads();
    }

    if (row < M && col < N) {
        C[N * row + col] = Csub;
    }
    
}

__global__ void matrixMultiShared2(float* A_p, float* B_p, float* C_p, int M, int N, int K) {
    __shared__ float subA[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float subB[BLOCK_SIZE][BLOCK_SIZE];
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    float sum_tmp = 0;
    for (int i = 0; i < (K / BLOCK_SIZE); i++) {
        // 计算每个block前先将数据从global memory读入shared memory
        subA[threadIdx.x][threadIdx.y] = A_p[row * K + (i * BLOCK_SIZE + threadIdx.y)];
        subB[threadIdx.x][threadIdx.y] = B_p[(i * BLOCK_SIZE + threadIdx.x) * N + col];
        __syncthreads();
        
        for (int j = 0; j < BLOCK_SIZE; j++) {
            sum_tmp = sum_tmp + subA[threadIdx.x][j] * subB[j][threadIdx.y];
        }
        __syncthreads();
    }
    if (row < M && col < K) {
        C_p[row * N + col] = sum_tmp;
    }
}

void checkResult(float* hostRef, float* deviceRef, const int num_to_check) {
    double diff = 1.0E-6;
    for (size_t i = 0; i < num_to_check; i++) {
        if (abs(hostRef[i] - deviceRef[i]) > diff) {
            printf("result check faild\n");
            printf("%f(hostRef[%ld] != %f(deviceRef[%ld]))", hostRef[i], i, deviceRef[i], i);
            return;
        }
    }
    printf("result check successfully\n");
}



int main(int argc, char* argv[]) {
    clock_t start = 0, finish = 0;
    float time;
    int Axy = M * K;
    int Bxy = K * N;
    int Cxy = M * N;

    float *h_A, *h_B, *hostRef, *deviceRef;
    h_A = (float*)malloc(Axy * sizeof(float));
    h_B = (float*)malloc(Bxy * sizeof(float)); 

    hostRef = (float*)malloc(Cxy * sizeof(float));
    deviceRef = (float*)malloc(Cxy * sizeof(float));
    memset(hostRef, 0, Cxy * sizeof(float));
    memset(deviceRef, 0, Cxy * sizeof(float));
    checkResult(hostRef, deviceRef, Cxy);

    initial(h_A, Axy);
    //printMatrix(h_A, M, K);
    initial(h_B, Bxy);
    //printMatrix(h_B, K, N);

    start = clock();
    multiMatrixOnHost(h_A, h_B, hostRef, M, K, N);
    finish = clock();
    time = (float)(finish - start) / CLOCKS_PER_SEC;
    
    printf("\n");
	printf("------------------------------------------------------------------------------------\n");
	printf("Computing matrix product using multiplicateMatrixOnHost \n");
	printf("Matrix_hostRef: (%d x %d)  CPU运行时间为: %lfs\n", M, N, time);

    float *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, Axy * sizeof(float));
    hipMalloc((void**)&d_B, Bxy * sizeof(float));
    hipMalloc((void**)&d_C, Cxy * sizeof(float));

    hipMemcpy(d_A, h_A, Axy * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, Bxy * sizeof(float), hipMemcpyHostToDevice);

    printf("\n\n");
	printf("------------------------------------------------------------------------------------\n");
	printf("Computing matrix product using multiplicateMatrixOnDevice \n");
    

    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid((N + dimBlock.x - 1) / dimBlock.x, (M + dimBlock.y - 1) / dimBlock.y);
    printf("block: (%d  %d  %d)\n", dimBlock.x, dimBlock.y, dimBlock.z);
    printf("grid:  (%d  %d  %d)\n", dimGrid.x, dimGrid.y, dimGrid.z);

    hipEvent_t gpustart, gpustop;
    // 未使用shared memory版本
    
    float elapsedTime = 0.0;
    hipEventCreate(&gpustart);
    hipEventCreate(&gpustop);
    hipEventRecord(gpustart, 0);
    multiMatrixOnDevice<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, M, K, N);
    // cudaDeviceSynchronize();
    hipEventRecord(gpustop, 0);
    hipEventSynchronize(gpustop);
    hipEventElapsedTime(&elapsedTime, gpustart, gpustop);
    hipEventDestroy(gpustart);
    hipEventDestroy(gpustop);

    hipMemcpy(deviceRef, d_C, Cxy * sizeof(float), hipMemcpyDeviceToHost);
    //printMatrix(deviceRef, M, N);
    checkResult(hostRef, deviceRef, Cxy);
    printf("Matrix_deviceRef: (%d×%d)  <<<(%d,%d),(%d,%d)>>>  GPU运行时间为：%fs\n",
                M, N, dimGrid.x, dimGrid.y, dimBlock.x, dimBlock.y, elapsedTime / 1000);

    // shared memory 版本
    printf("\n");
	printf("------------------------------------------------------------------------------------\n");
	printf("Computing matrix product using matrixMultiShared1 \n");
    
    elapsedTime = 0.0;
    hipEventCreate(&gpustart);
    hipEventCreate(&gpustop);
    hipEventRecord(gpustart, 0);
    matrixMultiShared1<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, M, N, K);
    // cudaDeviceSynchronize();
    hipEventRecord(gpustop, 0);
    hipEventSynchronize(gpustop);

    hipEventElapsedTime(&elapsedTime, gpustart, gpustop);
    hipEventDestroy(gpustart);
    hipEventDestroy(gpustop);

    hipMemcpy(deviceRef, d_C, Cxy * sizeof(float), hipMemcpyDeviceToHost);
    checkResult(hostRef, deviceRef, Cxy);
    printf("Matrix_deviceRef: (%d×%d)  <<<(%d,%d),(%d,%d)>>>  GPU运行时间为：%fs\n",
		M, N, dimGrid.x, dimGrid.y, dimBlock.x, dimBlock.y, elapsedTime / 1000);


    printf("\n");
	printf("------------------------------------------------------------------------------------\n");
	printf("Computing matrix product using matrixMultiShared2 \n");
    
    elapsedTime = 0.0;
    hipEventCreate(&gpustart);
    hipEventCreate(&gpustop);
    hipEventRecord(gpustart, 0);
    matrixMultiShared2<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, M, N, K);
    // cudaDeviceSynchronize();
    hipEventRecord(gpustop, 0);
    hipEventSynchronize(gpustop);

    hipEventElapsedTime(&elapsedTime, gpustart, gpustop);
    hipEventDestroy(gpustart);
    hipEventDestroy(gpustop);

    hipMemcpy(deviceRef, d_C, Cxy * sizeof(float), hipMemcpyDeviceToHost);
    checkResult(hostRef, deviceRef, Cxy);
    printf("Matrix_deviceRef: (%d×%d)  <<<(%d,%d),(%d,%d)>>>  GPU运行时间为：%fs\n",
		M, N, dimGrid.x, dimGrid.y, dimBlock.x, dimBlock.y, elapsedTime / 1000);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    
    free(h_A);
    free(h_B);
    free(hostRef);
    free(deviceRef);

    hipDeviceReset();

}
