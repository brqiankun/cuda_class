#include<hip/hip_runtime.h>

//matrix multiplication

//matrices are stored in row-major order
//M(row, col) = *(M.elements + row * M.width + col)
struct Matrix{
    int width;
    int height;
    float* elements;
};

//thread block size 
#define BLOCK_SIZE 16

//forward declaration of the matrix multiplication kernel
__global__ void MatMulKernel(const Matrix A, const Matrix B, Matrix C);

void MatMul(const Matrix A, const Matrix B, Matrix C) {
    //load A and B to device memory
    Matrix d_A;
    d_A.width = A.width;
    d_A.height = A.height;
    size_t size = A.width * A.height * sizeof(float);
    hipMalloc(&d_A.elements, size);
    hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);
    
    Matrix d_B;
    d_B.width = B.width;
    d_B.height = B.height;
    size = B.width * B.height * sizeof(float);
    hipMalloc(&d_B.elements, size);
    hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice);

    //allocate C in device memory
    Matrix d_C;
    d_C.width = C.width;
    d_C.height = C.height;
    size = C.width * C.height * sizeof(float);
    hipMalloc(&d_C.elements, size);

    //invoke kernel
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(B.width / dimBlock.x, A.height / dimBlock.y);
    MatMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);

    //read C from device memory
    hipMemcpy(C.elements, d_C.elements, size, hipMemcpyDeviceToHost);

    //free device memory
    hipFree(d_A.elements);
    hipFree(d_B.elements);
    hipFree(d_C.elements);

}

//Matrix multiplication kernel called by MatMul()
__global__ void MatMulKernel(Matrix A, Matrix B, Matrix C) {
    //Each thread computes one element of C
    //by accumulating results into Cvalue
    float Cvalue = 0;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    for(int e = 0; e < A.width; e++) {
        Cvalue += A.elements[row * A.width + e] * B.elements[col + e * B.width];
    }
    C.elements[row * C.width + col] = Cvalue;
}

